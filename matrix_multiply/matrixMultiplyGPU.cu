#include "hip/hip_runtime.h"
#include "matrixMultiplyGPU.cuh"
#include <stdlib.h>
#include <stdio.h>

#define cudaCheck(expr) \
    do { \
        hipError_t e = (expr); \
        if (e != hipSuccess) { \
            fprintf(stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString(e), __FILE__, __LINE__); \
            abort(); \
        } \
    } while (false)



__host__ void matrixMultiply_GPU(int N, const float* A, const float* B, float* C, int *arg, int argCount)
{
    memset(C, 0.0f, N * N * sizeof(float)); 
    
    int M_LEN = N * N; 

    float* d_A; 
    float* d_B; 
    float* d_C; 

    cudaCheck(hipMalloc(&d_A, sizeof(float) * M_LEN));
    cudaCheck(hipMalloc(&d_B, sizeof(float) * M_LEN)); 
    cudaCheck(hipMalloc(&d_C, sizeof(float) * M_LEN)); 

    cudaCheck(hipMemcpy(d_A, A, sizeof(float) * M_LEN, hipMemcpyHostToDevice)); 
    cudaCheck(hipMemcpy(d_B, B, sizeof(float) * M_LEN, hipMemcpyHostToDevice)); 
    cudaCheck(hipMemcpy(d_C, C, sizeof(float) * M_LEN, hipMemcpyHostToDevice)); 

    int NUM_THREADS = 32; 
    int NUM_BLOCKS = N / NUM_THREADS; 

    dim3 grid(NUM_BLOCKS, NUM_BLOCKS); 
    dim3 threads(NUM_THREADS, NUM_THREADS); 
    
    matrixMultiplyKernel_GPU<<<grid, threads>>>(N, d_A, d_B, d_C, 0, 0, 0); 

    cudaCheck(hipMemcpy(C, d_C, sizeof(float) * M_LEN, hipMemcpyDeviceToHost)); 

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C); 	
}

__global__ void matrixMultiplyKernel_GPU(int N, const float* A, const float* B, float* C, int flag0, int flag1, int flag2)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;  

    float sum = 0.0f; 

    if ((row < N) && (col < N)) {
        
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col]; 
        }

        C[row * N + col] = sum; 
    }
}

