#include "hip/hip_runtime.h"
#include "matrixMultiplyGPU.cuh"

__host__ void matrixMultiply_GPU(int N, const float* A, const float* B, float* C, int *arg, int argCount)
{
    int M_LEN = N * N; 

    float* d_A; 
    float* d_B; 
    float* d_C; 

    //allocate memory on the device 
    hipMalloc(&d_A, sizeof(float) * M_LEN);
    hipMalloc(&d_B, sizeof(float) * M_LEN); 
    hipMalloc(&d_C, sizeof(float) * M_LEN);

    //copy across arrays to the device 
    hipMemcpy(d_A, A, sizeof(float) * M_LEN, hipMemcpyHostToDevice); 
    hipMemcpy(d_B, B, sizeof(float) * M_LEN, hipMemcpyHostToDevice); 
    hipMemcpy(d_C, C, sizeof(float) * M_LEN, hipMemcpyHostToDevice); 

    int NUM_THREADS = 32; //32 * 32 is 1024 which is upper limit for threads
    //essentially one block per row and column  
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS; 

    dim3 grid(NUM_BLOCKS, NUM_BLOCKS); //use a 2D grid representation 
    dim3 threads(NUM_THREADS, NUM_THREADS); 
    
    matrixMultiplyKernel_GPU<<<grid, threads>>>(N, d_A, d_B, d_C, 0, 0, 0); 

    hipMemcpy(C, d_C, sizeof(float) * M_LEN, hipMemcpyDeviceToHost); 

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C); 	
}

__global__ void matrixMultiplyKernel_GPU(int N, const float* A, const float* B, float* C, int flag0, int flag1, int flag2)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;  

    //make sure we don't go over edge of memory 
    if ((row < N) && (col < N)) {
        float sum = 0.0f; 
        //each thread handles a single row and column multiplication 
        for (int k = 0; k < N; ++k) {
            sum += A[k * N + col] * B[row * N + k]; 
        }

        C[row * N + col] = sum; 
    }
}