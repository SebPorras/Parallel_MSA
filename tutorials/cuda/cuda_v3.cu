
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>
#include <cmath>

// This macro lets us output the file and line number
// if a CUDA error occurs.
#define cudaCheck(expr) \
    do { \
        hipError_t e = (expr); \
        if (e != hipSuccess) { \
            fprintf(stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString(e), __FILE__, __LINE__); \
            abort(); \
        } \
    } while (false)


// Works with any number of blocks.
// Requires shared memory space for num_threads doubles
__global__
void sum(const double* f, int N, double* out) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load from memory, serial sum if grid size < N
    double s = 0.;
    for (int i = idx; i < N; i += gridDim.x * blockDim.x)
        s += f[i];

    // Store result of serial sum in shared memory
    extern __shared__ double smem[];
    smem[tid] = s;

    // Perform reduction over shared memory
    for (int i = blockDim.x/2; i > 0; i /= 2) {
        __syncthreads();
        if (tid < i)
            smem[tid] += smem[tid + i];
    }
    if (tid == 0) out[blockIdx.x] = smem[0];
}

int main() {
    const double lower = 0.;
    const double upper = 1.;
    const int N = 1'000'000;

    const double dx = (upper - lower) / N;

    // Populate f(xi)
    double* f = (double*)malloc(sizeof(*f) * N);
    for (int i = 0; i < N; ++i) {
        double xi = lower + (i + 0.5) * dx;
        f[i] = 1. / (xi * xi + 1.);
    }

    // Enable 8 byte shared memory banks for conflict-free access to consecutive doubles
    cudaCheck(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

    // Set up working memory
    const int NUM_THREADS = 1024;
    int num_blocks = (N + NUM_THREADS - 1) / NUM_THREADS;
    double* d_f; cudaCheck(hipMalloc((void**)&d_f, sizeof(*d_f) * N));
    // Over-allocating the result array to leave space for multiple reduction steps
    double* d_result; cudaCheck(hipMalloc((void**)&d_result, sizeof(*d_result) * num_blocks * 2));

    // Copy f(xi) array to GPU
    cudaCheck(hipMemcpy(d_f, f, sizeof(*f) * N, hipMemcpyHostToDevice));

    // Begin timing
    auto start = std::chrono::high_resolution_clock::now();

    // Calculate integral

    // Initial reduction will give num_blocks values in d_result
    sum<<<num_blocks, NUM_THREADS, sizeof(*f)*NUM_THREADS>>>(d_f, N, d_result);

    // Keep applying reduction to d_result until only one block is needed
    // (so the result of that block will be the full sum of f).
    // Uses extra space allocated in d_result to store the result of
    // the next reduction step (buf2).
    double* buf1 = d_result;
    double* buf2 = &d_result[((num_blocks+3)/4) * 4]; // Make sure buf2 is aligned to 256 bits
    while (num_blocks > 1) {
        int old_blocks = num_blocks; // This will be the number of elements we need to reduce over
        num_blocks = (num_blocks + NUM_THREADS - 1) / NUM_THREADS;
        sum<<<num_blocks, NUM_THREADS, sizeof(*f)*NUM_THREADS>>>(buf1, old_blocks, buf2);
        double* tmp = buf2;
        buf2 = buf1;
        buf1 = tmp;
    }

    // Copy final result back to the host
    double integral;
    cudaCheck(hipMemcpy(&integral, buf1, sizeof(integral), hipMemcpyDeviceToHost));
    integral *= dx;

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::micro> duration_us = end - start;

    printf("Integral was %.15g\n", integral);
    printf("Error was %.10e\n", std::abs(integral - M_PI/4.));
    printf("Time taken: %g us\n", duration_us.count());

    hipFree(d_f);
    hipFree(d_result);
    free(f);
}
